#include "hip/hip_runtime.h"
//#include <stdio.h>
#include <stdlib.h>
#include "matrix_gpu.h"

extern int block_size;

__global__ void calc_sum_rows_gpu(float *A_vals, float *b, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i;

	if(idx<n){
		b[idx]=0.0;
		for(i=0;i<m;i++){
			if(A_vals[i + idx*m] > 0.0)
				b[idx] += A_vals[i + idx*m];
			else
				b[idx] -= A_vals[i + idx*m];
		}
	}
}

__global__ void calc_sum_cols_gpu(float *A_vals, float *b, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i;

	if(idx<m){
		b[idx] = 0.0;
		for(i=0;i<n;i++){
			if(A_vals[idx + i*m] > 0.0)
				b[idx] += A_vals[idx + i*m];
			else
				b[idx] -= A_vals[idx + i*m];
		}
	}
}
	
__global__ void calc_vec_reduce_gpu(float *vec, float *b, int n){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(idx<n/2)
		b[idx] = vec[2*idx] + vec[(2*idx)+1];
	if(n%2!=0 && idx == (n/2)-1)
		b[idx] += vec[(2*idx) + 2];
}

extern int sum_rows_gpu(float *A_vals, float *row, int n, int m){
	float *A_vals_d, *row_d;

	//row = (float*)calloc(n,sizeof(float));

	hipMalloc( (void**)&A_vals_d, n*m*sizeof(float));
	hipMalloc( (void**)&row_d, n*sizeof(float));

	hipMemcpy(A_vals_d, A_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 dimBlock(block_size);
	dim3 dimGrid((n/dimBlock.x) + (!(n%dimBlock.x)?0:1));

	calc_sum_rows_gpu <<<dimGrid,dimBlock>>> (A_vals_d, row_d, n, m);

	hipMemcpy(row, row_d, n*sizeof(float), hipMemcpyDeviceToHost);
	
	//free(row);
	hipFree(A_vals_d); hipFree(row_d);
	
	//printf("TESTING ROW FUNCTION CALL\n");
	return 0;
}

extern int sum_cols_gpu(float *A_vals, float *col, int n, int m){
	float *A_vals_d, *col_d;

	//col = (float*)calloc(m,sizeof(float));

	hipMalloc( (void**)&A_vals_d, n*m*sizeof(float));
	hipMalloc( (void**)&col_d, m*sizeof(float));

	hipMemcpy(A_vals_d, A_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 dimBlock(block_size);
	dim3 dimGrid((m/dimBlock.x) + (!(m%dimBlock.x)?0:1));

	calc_sum_cols_gpu <<<dimGrid,dimBlock>>> (A_vals_d, col_d, n, m);

	hipMemcpy(col, col_d, m*sizeof(float), hipMemcpyDeviceToHost);
	
	//free(col);
	hipFree(A_vals_d); hipFree(col_d);
	
	//printf("TESTING COL FUNCTION CALL\n");
	return 0;
}

extern float vec_reduce_gpu(float *vec, int n, float* sum2){
	float sum, *vec_d;

	hipMalloc( (void**)&vec_d, n*sizeof(float));

	hipMemcpy(vec_d, vec, n*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 dimBlock(block_size);
	dim3 dimGrid((n/dimBlock.x) + (!(n%dimBlock.x)?0:1));

	while(n > 1){
		calc_vec_reduce_gpu <<<dimGrid,dimBlock>>> (vec_d, vec_d, n);
		n /= 2;
	}
	hipMemcpy(&sum, &vec_d[0], sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(vec_d);
	*sum2 = sum;
	//printf("total sum = %f\n", *sum2);
	//printf("TESTING ROW FUNCTION CALL\n");
	return sum;
}
