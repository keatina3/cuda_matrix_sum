#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include "matrix_gpu.h"

//extern int block_size;

__global__ void calc_sum_rows_gpu(float *A_vals, float *b, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i;

	if(idx<n){
		b[idx]=0.0;
		for(i=0;i<m;i++){
			if(A_vals[i + idx*m] > 0.0)
				b[idx] += A_vals[i + idx*m];
			else
				b[idx] -= A_vals[i + idx*m];
		}
	}
}

__global__ void calc_sum_cols_gpu(float *A_vals, float *b, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i;

	if(idx<m){
		b[idx] = 0.0;
		for(i=0;i<n;i++){
			if(A_vals[idx + i*m] > 0.0)
				b[idx] += A_vals[idx + i*m];
			else
				b[idx] -= A_vals[idx + i*m];
		}
	}
}
	
__global__ void calc_vec_reduce_gpu(float *vec, float *b, int n){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(idx<n/2)
		b[idx] = vec[2*idx] + vec[(2*idx)+1];
	if(n%2!=0 && idx == (n/2)-1)
		b[idx] += vec[(2*idx) + 2];
}

extern void sum_rows_gpu(float *A_vals, float *row, int block_size, int n, int m, float *tau){
	float *A_vals_d, *row_d;
	struct timeval start, end;

	hipMalloc( (void**)&A_vals_d, n*m*sizeof(float));
	hipMalloc( (void**)&row_d, n*sizeof(float));

	hipMemcpy(A_vals_d, A_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 dimBlock(block_size);
	dim3 dimGrid((n/dimBlock.x) + (!(n%dimBlock.x)?0:1));

	gettimeofday(&start,NULL);
	calc_sum_rows_gpu <<<dimGrid,dimBlock>>> (A_vals_d, row_d, n, m);
	gettimeofday(&end,NULL);
	
	*tau = (float)(end.tv_sec-start.tv_sec) + (float)(end.tv_usec - start.tv_usec)/(1E06);

	hipMemcpy(row, row_d, n*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(A_vals_d); hipFree(row_d);
}

extern void sum_cols_gpu(float *A_vals, float *col, int block_size, int n, int m, float *tau){
	float *A_vals_d, *col_d;
	struct timeval start, end;

	hipMalloc( (void**)&A_vals_d, n*m*sizeof(float));
	hipMalloc( (void**)&col_d, m*sizeof(float));
	
	hipMemcpy(A_vals_d, A_vals, n*m*sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(block_size);
	dim3 dimGrid((m/dimBlock.x) + (!(m%dimBlock.x)?0:1));

	gettimeofday(&start,NULL);
	calc_sum_cols_gpu <<<dimGrid,dimBlock>>> (A_vals_d, col_d, n, m);
	gettimeofday(&end,NULL);
	
	*tau = (float)(end.tv_sec-start.tv_sec) + (float)(end.tv_usec - start.tv_usec)/(1E06);
	
	hipMemcpy(col, col_d, m*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(A_vals_d); hipFree(col_d);
}

extern void vec_reduce_gpu(float *vec, int block_size, int n, float* sum, float *tau){
	float *vec_d;
	struct timeval start, end;

	hipMalloc( (void**)&vec_d, n*sizeof(float));

	hipMemcpy(vec_d, vec, n*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 dimBlock(block_size);
	dim3 dimGrid((n/dimBlock.x) + (!(n%dimBlock.x)?0:1));

	gettimeofday(&start,NULL);
	while(n > 1){
		calc_vec_reduce_gpu <<<dimGrid,dimBlock>>> (vec_d, vec_d, n);
		n /= 2;
	}
	gettimeofday(&end,NULL);
	
	*tau = (float)(end.tv_sec-start.tv_sec) + (float)(end.tv_usec - start.tv_usec)/(1E06);
	
	hipMemcpy(sum, &vec_d[0], sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(vec_d);
}
